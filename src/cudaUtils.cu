/*
 * cudaUtils
 *
 * Evan Herbst
 * 3 / 21 / 12
 */

#include <cstdio>
#include "cuda_util/cudaUtils.h"

void cudaCall(const hipError_t err, const char* filename, const int line)
{
	const char* s = hipGetErrorString(err);
	if(err != hipSuccess)
		printf("cuda error at %s:%d: %s\n", filename, line, s);
}
